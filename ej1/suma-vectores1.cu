
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 500

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void VecAdd(int* DA, int* DB, int* DC)
{
	int i = threadIdx.x;
    DC[i] = DA[i] + DB[i];
}

int main()
{ int HA[N], HB[N], HC[N];
  int *DA, *DB, *DC;
  int i; int size = N*sizeof(int);
  float ms, msa, msb, msc;
  hipEvent_t startEvent, stopEvent, totalStart, totalEnd;
  checkCuda( hipEventCreate(&startEvent));
  checkCuda( hipEventCreate(&totalStart));
  checkCuda( hipEventCreate(&stopEvent));
  checkCuda( hipEventCreate(&totalEnd));
  
  checkCuda( hipEventRecord(totalStart, 0));

  // reservamos espacio en la memoria global del device
  checkCuda ( hipEventRecord(startEvent, 0));
  hipMalloc((void**)&DA, size);
  checkCuda( hipEventRecord(stopEvent, 0));
  checkCuda( hipEventSynchronize(stopEvent));
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("cudaMallod DA :  (ms) %f\n", ms);

  checkCuda ( hipEventRecord(startEvent, 0));
  hipMalloc((void**)&DB, size);
  checkCuda( hipEventRecord(stopEvent, 0));
  checkCuda( hipEventSynchronize(stopEvent));
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("cudaMallod DB :  (ms) %f\n", ms);

  checkCuda ( hipEventRecord(startEvent, 0));
  hipMalloc((void**)&DC, size);
  checkCuda( hipEventRecord(stopEvent, 0));
  checkCuda( hipEventSynchronize(stopEvent));
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("cudaMallod DC :  (ms) %f\n", ms);
  
  // inicializamos HA y HB
  for (i=0; i<N; i++) {HA[i]=-i; HB[i] = 3*i;}
  
  // copiamos HA y HB del host a DA y DB en el device, respectivamente
  checkCuda ( hipEventRecord(startEvent, 0));
  hipMemcpy(DA, HA, size, hipMemcpyHostToDevice);
  checkCuda( hipEventRecord(stopEvent, 0));
  checkCuda( hipEventSynchronize(stopEvent));
  checkCuda( hipEventElapsedTime(&msa, startEvent, stopEvent));
  printf("cudaMemCPY HA:  (ms) %f\n", msa);

  checkCuda ( hipEventRecord(startEvent, 0));
  hipMemcpy(DB, HB, size, hipMemcpyHostToDevice);
  checkCuda( hipEventRecord(stopEvent, 0));
  checkCuda( hipEventSynchronize(stopEvent));
  checkCuda( hipEventElapsedTime(&msb, startEvent, stopEvent));
  printf("cudaMemCPY HB:  (ms) %f\n", msb);
  
  // llamamos al kernel (1 bloque de N hilos)
  checkCuda ( hipEventRecord(startEvent, 0));
  VecAdd <<<1, N>>>(DA, DB, DC);	// N hilos ejecutan el kernel en paralelo
  checkCuda( hipEventRecord(stopEvent, 0));
  checkCuda( hipEventSynchronize(stopEvent));
  checkCuda( hipEventElapsedTime(&msc, startEvent, stopEvent));
  printf("Kernel execution time:  (ms) %f\n", msc);
  

  checkCuda ( hipEventRecord(startEvent, 0));
  // copiamos el resultado, que está en la memoria global del device, (DC) al host (a HC)
  hipMemcpy(HC, DC, size, hipMemcpyDeviceToHost);
  checkCuda( hipEventRecord(stopEvent, 0));
  checkCuda( hipEventSynchronize(stopEvent));
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("cudaMemCPY HC:  (ms) %f\n", ms);
  
  // liberamos la memoria reservada en el device
  hipFree(DA); hipFree(DB); hipFree(DC);  
  
  // una vez que tenemos los resultados en el host, comprobamos que son correctos
  for (i = 0; i < N; i++) // printf("%d + %d = %d\n",HA[i],HB[i],HC[i]);
    if (HC[i]!= (HA[i]+HB[i])) 
		{printf("error en componente %d\n", i); break;}

  checkCuda( hipEventRecord( totalEnd, 0));
  checkCuda( hipEventSynchronize(totalEnd));
  checkCuda( hipEventElapsedTime(&ms, totalStart, totalEnd));
  printf("Time for sequential transfer and execute (ms): %f\n", ms);

  printf("Bandwith in the transfer DA <-- HA %f GBs\t\n", size/(msa/1000)/1000000000);
  printf("Bandwith in the transfer DB <-- HB %f GBs\t\n", size/(msb/1000)/1000000000);
  printf("Bandwith in the transfer DC <-- HC %f GBs\t\n", size/(msc/1000)/1000000000);

  printf("Finished\t\n");
  return 0;
} 
